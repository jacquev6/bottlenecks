// Copyright 2022 Vincent Jacques


#include <hip/hip_runtime.h>
#include <cstdlib>


#define BLOCK_SIZE 512

__global__ void kernel(double* x, const int repetitions) {
  x[threadIdx.x] = 3.14;
  for (int j = 0; j != repetitions; ++j) {
    x[threadIdx.x] *= j;
  }
}

int main(int argc, char* argv[]) {
  if (argc < 2) exit(1);
  const int multiplier = std::atoi(argv[1]);
  if (multiplier < 1 || multiplier > 1024) exit(1);
  const int repetitions = 1024 * 1024 * multiplier;

  double* d_x;
  hipMalloc(&d_x, BLOCK_SIZE * sizeof(double));
  kernel<<<1, BLOCK_SIZE>>>(d_x, repetitions);
  double x[BLOCK_SIZE];
  hipMemcpy(x, d_x, BLOCK_SIZE * sizeof(double), hipMemcpyDeviceToHost);
  hipFree(d_x);
}
